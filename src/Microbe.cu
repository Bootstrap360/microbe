#include "hip/hip_runtime.h"

#include <stdio.h>
#include <thrust/sequence.h>

#include "Microbe.h"

__host__ Microbe::Microbe(int ID, double dt, int num_poses, int num_instructions)
    : m_ID(ID)
{
    // h_poses = thrust::host_vector<State> (num_poses);
    // h_velocities = thrust::host_vector<Velocity> (num_poses);
    thrust::host_vector<int> h_instructions (num_instructions);
    for(int i = 0; i < num_instructions; i ++)
    {
        h_instructions[i] = i;
        printf("generating %i ", m_ID);
        printf("instruction = %d\n", h_instructions[i]);
    }
    thrust::device_vector<int> d_instructions = h_instructions;
    thrust::device_ptr<int> dev_ptr = d_instructions.data();
    d_instructions_ptr = thrust::raw_pointer_cast(dev_ptr);
    d_instructions_length = num_instructions;

}

// TODO: d_instructions is going to go out of scope. Need to hold onto memory

// __host void Microbe::Upload()
// {
//     d_poses = h_poses;
//     d_velocities = h_velocities;
//     d_instructions = h_instructions;
// }

//  __device__ void Microbe::Step(const State& pose, 
//                                     const State& velocity, 
//                                     const Command& command, 
//                                     State& nextPose, 
//                                     State& nextVelocity)
// {

// }

__device__ void Microbe::Simulate()
{
    for(int i = 0; i < d_instructions_length; i ++)
    {
         printf("Simulating %i ",m_ID);
         printf("instruction = %i \n", d_instructions_ptr[i]);
    }
}

// __device__ Microbe&  Microbe::AsexualReproduce()
// {
//     printf("AsexualReproduce %d", m_ID);
//     return Microbe(m_ID, m_dt);
// }

// __device__ Microbe&  Microbe::SexualReproduce( const Microbe & other)
// {
//     printf("SexualReproduce %d  and %d ", m_ID, other.m_ID);
//     return Microbe(m_ID + other.m_ID);
// }


// __device bool Microbe::operator< (const Microbe & rhs) const
// {
//     return false;
// }

// __global__ 
// void kernal_Simulate(int num_microbes)
// {
//     int microbe_number = blockIdx.x * blockDim.x + threadIdx.x;
//     if(microbe_number < num_microbes)
//     {
//             printf("Hello from block %d, blockdim %d, thread %d\n", blockIdx.x,  blockDim.x, threadIdx.x);
//             printf("Calling simulate on microbe_number %d\n", microbe_number);
//         //     d_microbes[microbe_number].Simulate();
//     }
// }

