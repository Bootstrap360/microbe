#include "hip/hip_runtime.h"

#include <stdio.h>

#include "Microbe.h"

__host__ Microbe::Microbe(long ID, double dt, int num_poses, int num_instructions)
    : m_ID(ID)
{
    // h_poses = thrust::host_vector<State> (num_poses);
    // h_velocities = thrust::host_vector<Velocity> (num_poses);
    // d_instructions = thrust::host_vector<Velocity> (num_poses);
}

// __host void Microbe::Upload()
// {
//     d_poses = h_poses;
//     d_velocities = h_velocities;
//     d_instructions = h_instructions;
// }

//  __device__ void Microbe::Step(const State& pose, 
//                                     const State& velocity, 
//                                     const Command& command, 
//                                     State& nextPose, 
//                                     State& nextVelocity)
// {

// }

__device__ void Microbe::Simulate()
{
    printf("Simulating %d\n", m_ID);
}

// __device__ Microbe&  Microbe::AsexualReproduce()
// {
//     printf("AsexualReproduce %d", m_ID);
//     return Microbe(m_ID, m_dt);
// }

// __device__ Microbe&  Microbe::SexualReproduce( const Microbe & other)
// {
//     printf("SexualReproduce %d  and %d ", m_ID, other.m_ID);
//     return Microbe(m_ID + other.m_ID);
// }


// __device bool Microbe::operator< (const Microbe & rhs) const
// {
//     return false;
// }

// __global__ 
// void kernal_Simulate(int num_microbes)
// {
//     int microbe_number = blockIdx.x * blockDim.x + threadIdx.x;
//     if(microbe_number < num_microbes)
//     {
//             printf("Hello from block %d, blockdim %d, thread %d\n", blockIdx.x,  blockDim.x, threadIdx.x);
//             printf("Calling simulate on microbe_number %d\n", microbe_number);
//         //     d_microbes[microbe_number].Simulate();
//     }
// }

