#include "hip/hip_runtime.h"

#include <stdio.h>
#include <thrust/sequence.h>
// #include <uniform_real_distribution.h>
#include <thrust/random.h>

#include "Microbe.h"

struct GenRand
{
    __device__
    float operator () (int idx)
    {
        thrust::default_random_engine randEng;
        thrust::uniform_real_distribution<float> uniDist;
        randEng.discard(idx);
        return uniDist(randEng);
    }
};

__host__ Microbe::Microbe(int ID, double dt, int num_poses, int num_instructions)
    : m_ID(ID)
{
    // h_poses = thrust::host_vector<State> (num_poses);
    // h_velocities = thrust::host_vector<Velocity> (num_poses);

    
    h_instructions = thrust::host_vector<double> (num_instructions);

    d_instructions = h_instructions;
    thrust::transform(
        thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(num_instructions),
        d_instructions.begin(),
        GenRand());
}

// TODO: d_instructions is going to go out of scope. Need to hold onto memory

__host__ void Microbe::Upload()
{
    // d_poses = h_poses;
    // d_velocities = h_velocities;
    d_instructions = h_instructions;
}

__host__ void Microbe::Download()
{
    // h_poses = d_poses;
    // h_velocities = d_velocities;
    h_instructions = d_instructions;
}

__host__ MicrobeData Microbe::GetGPUData()
{
    MicrobeData data;
    data.instructions = convertToKernel(d_instructions);
    data.ID = m_ID;
    return data;
}

//  __device__ void Microbe::Step(const State& pose, 
//                                     const State& velocity, 
//                                     const Command& command, 
//                                     State& nextPose, 
//                                     State& nextVelocity)
// {

// }

// __global__ void Microbe::kernel_Simulate()
// {
//     printf("Hello from block %d, blockdim %d, thread %d\n", blockIdx.x,  blockDim.x, threadIdx.x);
// }
__host__ void Microbe::Simulate()
{

    kernel_Simulate<<< 1,1 >>>(m_ID, convertToKernel(d_instructions));
}

// __device__ void Microbe::Simulate()
// {
//     for(int i = 0; i < d_instructions_length; i ++)
//     {
//          printf("Simulating %i ",m_ID);
//          printf("instruction = %i \n", d_instructions_ptr[i]);
//     }
// }

// __device__ Microbe&  Microbe::AsexualReproduce()
// {
//     printf("AsexualReproduce %d", m_ID);
//     return Microbe(m_ID, m_dt);
// }

// __device__ Microbe&  Microbe::SexualReproduce( const Microbe & other)
// {
//     printf("SexualReproduce %d  and %d ", m_ID, other.m_ID);
//     return Microbe(m_ID + other.m_ID);
// }


// __device bool Microbe::operator< (const Microbe & rhs) const
// {
//     return false;
// }

__global__
void kernel_Simulate(int ID, KernelArray<double> instructions)
{
    printf("Hello from block %d, blockdim %d, thread %d\n", blockIdx.x,  blockDim.x, threadIdx.x);
    printf("num instructions %d\n", instructions.size);
    for(int instruction_ptr = 0; instruction_ptr < instructions.size; instruction_ptr ++)
    {
        printf("%d, %d, %f\n", ID, instruction_ptr, instructions.array[instruction_ptr]);
    }
    
}

 


// __global__ 
// void kernal_Simulate(int num_microbes)
// {
//     int microbe_number = blockIdx.x * blockDim.x + threadIdx.x;
//     if(microbe_number < num_microbes)
//     {
//             printf("Hello from block %d, blockdim %d, thread %d\n", blockIdx.x,  blockDim.x, threadIdx.x);
//             printf("Calling simulate on microbe_number %d\n", microbe_number);
//         //     d_microbes[microbe_number].Simulate();
//     }
// }

