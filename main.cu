#include "hip/hip_runtime.h"

#include <thrust/version.h>

#include <thrust/device_vector.h> 
#include <thrust/host_vector.h> 
#include <thrust/transform.h> 
#include <thrust/sequence.h> 
#include <thrust/copy.h> 
#include <thrust/fill.h> 
#include <thrust/replace.h> 
#include <thrust/functional.h> 
#include <iostream>

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Microbe.h"

struct printf_functor
{
  __host__ __device__
  void operator()(double x)
  {
    printf("%f\n", x);
  }
};

__global__
void hellWorld(int test)
{
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main(int argc, char *argv[])
{
    int major = THRUST_MAJOR_VERSION;
    int minor = THRUST_MINOR_VERSION;

    std::cout << "Thrust v" << major << "." << minor << std::endl;
    
    double dt = 0.1;
    int num_microbes = 8;

    // Microbe m(0, 0.1);

    thrust::host_vector<Microbe> h_microbes;
    for(int i = 0; i < num_microbes; i++)
    {
        Microbe newMicrobe(i, dt);
        h_microbes.push_back(newMicrobe);
    }

    thrust::device_vector<Microbe> d_microbes = h_microbes;

    thrust::for_each(d_microbes.begin(), d_microbes.end(), Simulate_functor());

    // kernal_Simulate <<<1, 32>>>(d_microbes.begin(), d_microbes.end());
    // kernal_Simulate <<<5, 32>>>(num_microbes);
    // hipDeviceSynchronize();
    
    return 0;
}