#include "hip/hip_runtime.h"

#include <thrust/version.h>

#include <thrust/device_vector.h> 
#include <thrust/host_vector.h> 
#include <thrust/transform.h> 
#include <thrust/sequence.h> 
#include <thrust/copy.h> 
#include <thrust/fill.h> 
#include <thrust/replace.h> 
#include <thrust/functional.h> 
#include <iostream>

#include "Microbe.h"

struct printf_functor
{
  __host__ __device__
  void operator()(double x)
  {
    printf("%f\n", x);
  }
};

__global__
void hellWorld(int test)
{
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main(int argc, char *argv[])
{
    int major = THRUST_MAJOR_VERSION;
    int minor = THRUST_MINOR_VERSION;

    std::cout << "Thrust v" << major << "." << minor << std::endl;
    
    // double dt = 0.1;
    // int num_microbes = 8;

    // Microbe m(0, 0.1);

    // thrust::host_vector<Microbe> h_microbes();
    // for(int i = 0; i < num_microbes; i++)
    // {
    //     Microbe newMicrobe(i, dt);
    //     host_vector.push_back(newMicrobe);
    // }

    // thrust::device_vector<Microbe> d_microbes = h_microbes;

    // kernal_Simulate <<1, 32>>(d_microbes.begin(), d_microbes.end());
    hellWorld<<<1,32>>>(0.5);

    return 0;
}