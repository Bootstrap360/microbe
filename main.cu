#include "hip/hip_runtime.h"

#include <thrust/version.h>

#include <vector>
#include <iostream>

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "Microbe.h"

struct printf_functor
{
  __host__ __device__
  void operator()(double x)
  {
    printf("%f\n", x);
  }
};

__global__
void hellWorld(int test)
{
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main(int argc, char *argv[])
{
    int major = THRUST_MAJOR_VERSION;
    int minor = THRUST_MINOR_VERSION;

    std::cout << "Thrust v" << major << "." << minor << std::endl;
    
    double dt = 0.1;
    int num_microbes = 8;

    // Microbe m(0, 0.1);

    std::vector<Microbe> microbes;
    for(int i = 0; i < num_microbes; i++)
    {
        Microbe newMicrobe(i, dt);
        microbes.push_back(newMicrobe);
    }

    thrust::host_vector<MicrobeData> h_microbesData;
    for(int i = 0; i < num_microbes; i++)
    {
        h_microbesData.push_back(microbes[i].GetGPUData());
    }
    thrust::device_vector<MicrobeData> d_microbesData;
    d_microbesData = h_microbesData;


    // thrust::device_vector<Microbe> d_microbes = microbes;

    thrust::for_each(d_microbesData.begin(), d_microbesData.end(), functor_Simulate());

    // kernal_Simulate <<<1, 32>>>(d_microbes.begin(), d_microbes.end());
    // kernal_Simulate <<<5, 32>>>(num_microbes);
    // hipDeviceSynchronize();
    
    return 0;
}